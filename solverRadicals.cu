#include "hip/hip_runtime.h"
/*
Compile and run inside a Google Colab notebook with a GPU using the commands:

!nvidia-smi
!nvcc  -o diffusion -x cu -lnvToolsExt drive/MyDrive/path/to/file/diffusionCUDARevised.cu
!./diffusion

The first line is not strictly necesary, but it lets us check what GPU we have,
probably a Tesla T4.

The second line runs the compiller. It is recomended you put this file on your
Google Drive and mount your drive to the Colab session.

The third line runs the code. Every 100 steps it will print out the current step
number.
*/

#include <string>
#include <math.h>
#include <iostream>
#include <fstream>
#include <cstdlib>
#include <vector>
#include <nvToolsExt.h>
#include "CLI/CLI.hpp"
#include <stdio.h>
#include <ctime>

// Use a struct for the configuration to make it easier to pass around
struct Config
{
    float diffCoeff;
    float radFormRate;
    float k1;
    float k2;
    float doseRate;
    int irrTime;
    int dimT;
    int dimX;
    int dimY;
    int dimZ;
    int DSIZE;
    int SSIZE;
    std::string outputFileNamePrefix;
};

#define blocks 80    // Should be number of streaming multiprocessors x2
#define threads 1024 // Should probably be 1024

#define cudaCheckErrors(msg)                                   \
    do                                                         \
    {                                                          \
        hipError_t __err = hipGetLastError();                \
        if (__err != hipSuccess)                              \
        {                                                      \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                    msg, hipGetErrorString(__err),            \
                    __FILE__, __LINE__);                       \
            fprintf(stderr, "*** FAILED - ABORTING\n");        \
            exit(1);                                           \
        }                                                      \
    } while (0)

__global__ void finiteDiff(const float *inputVal, float *outputVal,
                           const float *inputRad, float *outputRad,
                           float *saveSlice, float *saveActivity,
                           Config *config, int tStamp)
{

    for (int index = threadIdx.x + blockDim.x * blockIdx.x; index < config->DSIZE; index += gridDim.x * blockDim.x)
    {

        // Determine array location
        int x = index % config->dimX;
        int y = ((index - x) / config->dimX) % config->dimY;
        int z = (index - x - y * config->dimX) / config->dimY / config->dimX;
        float radicalLoss = 0;
        float crossLinking = 0;
        float irradiationOn = 1;
        if (tStamp > config->irrTime)
        {
            irradiationOn = 0;
        }

        // Assuming not a boundary of the array
        if (x > 0 && y > 0 && z > 0 && x < config->dimX - 1 && y < config->dimY - 1 && z < config->dimZ - 1)
        {
            // Oxygen concentration - initial condition
            outputVal[index] = inputVal[index];

            // Applying the laplacian for oxygen diffusion across the 3 dimensions
            outputVal[index] += config->diffCoeff * (inputVal[index - 1] + inputVal[index + 1] - 2 * inputVal[index]);
            outputVal[index] += config->diffCoeff * (inputVal[index - config->dimX] + inputVal[index + config->dimX] - 2 * inputVal[index]);
            outputVal[index] += config->diffCoeff * (inputVal[index - config->dimX * config->dimY] + inputVal[index + config->dimX * config->dimY] - 2 * inputVal[index]);

            // Radical concentration - initial condition + radical formation
            outputRad[index] = inputRad[index] + config->radFormRate * config->doseRate * irradiationOn;

            // Radical oxidation calculation
            // Need to account for zero concentration cases
            // Assume that the radical loss consumes fully the lowest quantity
            radicalLoss = config->k2 * outputRad[index] * outputVal[index];
            if (radicalLoss > outputRad[index])
            {
                if (outputVal[index] > outputRad[index])
                {
                    radicalLoss = outputRad[index];
                }
                else
                {
                    radicalLoss = outputVal[index];
                }
            }
            else if (radicalLoss > outputVal[index])
            {
                radicalLoss = outputVal[index];
            }

            // Apply radical loss
            outputRad[index] -= radicalLoss;
            outputVal[index] -= radicalLoss;

            // Crosslinking
            crossLinking = config->k1 * outputRad[index] * outputRad[index];
            // Boundary conditions - this quantity cannot go below zero
            if (crossLinking > outputRad[index])
            {
                outputRad[index] = 0;
            }
            else
            {
                // Normal case
                outputRad[index] -= crossLinking;
            }
        }
        else
        {
            // Fixed concentration outside
            outputVal[index] = inputVal[index];
        }

        if (x == (int)config->dimX / 2)
        {
            saveSlice[tStamp * config->dimY * config->dimZ + y + z * config->dimY] = outputVal[index];
            saveActivity[tStamp * config->dimY * config->dimZ + y + z * config->dimY] = radicalLoss;
        }
    }
}

// Take one step of the algorithm
void stepAlgo(float *d_inputVal, float *d_outputVal, float *d_inputRad,
              float *d_outputRad, float *d_saveSlice, float *d_saveActivity, int tStamp, Config *config)
{

    // Run the main algorithm
    finiteDiff<<<blocks, threads>>>(d_inputVal, d_outputVal, d_inputRad,
                                    d_outputRad, d_saveSlice, d_saveActivity,
                                    config, tStamp);
    cudaCheckErrors("main kernel launch failure");
    hipDeviceSynchronize();
}

void run(float *inputVal, float *inputRad, float *saveSlice, float *saveActivity, Config *config)
{
    int counter = 1;

    // Declare device pointers
    float *d_inputVal;
    float *d_outputVal;
    float *d_inputRad;
    float *d_outputRad;
    float *d_saveSlice;
    float *d_saveActivity;

    // Allocate memory on the gpu
    hipMalloc(&d_inputVal, config->DSIZE * sizeof(float));
    hipMalloc(&d_outputVal, config->DSIZE * sizeof(float));
    hipMalloc(&d_inputRad, config->DSIZE * sizeof(float));
    hipMalloc(&d_outputRad, config->DSIZE * sizeof(float));
    hipMalloc(&d_saveSlice, config->SSIZE * sizeof(float));
    hipMalloc(&d_saveActivity, config->SSIZE * sizeof(float));
    cudaCheckErrors("hipMalloc failure"); // error checking

    // Copy data to the GPU
    hipMemcpy(d_inputVal, inputVal, config->DSIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_outputVal, inputVal, config->DSIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_inputRad, inputRad, config->DSIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_outputRad, inputRad, config->DSIZE * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckErrors("hipMemcpy H2D failure");

    std::time_t msStart = std::time(nullptr);
    while (counter <= config->dimT)
    {
        // Print out the counter every 1000 iterations
        if ((counter - 1) % 1000 == 0)
        {
            std::cout << counter - 1 << std::endl;
        }
        // Run 1 step of the algorithm
        stepAlgo(d_inputVal, d_outputVal, d_inputRad, d_outputRad, d_saveSlice, d_saveActivity, counter - 1, config);
        counter++;

        // Run another step but with the input and output arrays flipped so
        // the memory doesn't need copied
        stepAlgo(d_outputVal, d_inputVal, d_outputRad, d_inputRad, d_saveSlice, d_saveActivity, counter - 1, config);
        counter++;
    }
    std::time_t msEnd = std::time(nullptr);

    // Give timing information
    std::cout << double(msEnd - msStart) * double(1000) / double(counter) << " ms per step\n";

    // Copy data off the GPU
    hipMemcpy(inputVal, d_inputVal, config->DSIZE * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(saveSlice, d_saveSlice, config->SSIZE * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(saveActivity, d_saveActivity, config->SSIZE * sizeof(float), hipMemcpyDeviceToHost);

    cudaCheckErrors("kernel execution failure or hipMemcpy H2D failure");

    // Free the memory on the GPU
    hipFree(d_inputVal);
    hipFree(d_outputVal);
    hipFree(d_inputRad);
    hipFree(d_outputRad);
    hipFree(d_saveSlice);
    hipFree(d_saveActivity);
}

int main(int argc, char **argv)
{
    CLI::App app{"Radical diffusion simulation"};
    Config config;

    // Declare the command line options
    config.diffCoeff = 0.1;
    app.add_option("--diffCoeff", config.diffCoeff, "Diffusion coefficient", (bool)true);
    config.radFormRate = 0.00025;
    app.add_option("--radFormRate", config.radFormRate, "Radical formation rate", (bool)true);
    config.k1 = 0.001;
    app.add_option("--k1", config.k1, "Rate of crosslinking", (bool)true);
    config.k2 = 1;
    app.add_option("--k2", config.k2, "Rate of radical oxidation", (bool)true);
    config.doseRate = 700;
    app.add_option("--doseRate", config.doseRate, "Dose rate", (bool)true);
    config.irrTime = 10000;
    app.add_option("--irrTime", config.irrTime, "Irradiation time", (bool)true);
    config.dimT = 20000;
    app.add_option("--totalTime", config.dimT, "Total time", (bool)true);
    std::vector<int> dimXYZ = {100, 100, 500};
    app.add_option("--dimXYZ", dimXYZ, "Dimensions X Y Z of the array", (bool)true)->expected(3);

    CLI11_PARSE(app, argc, argv);

    config.dimX = dimXYZ[0];
    config.dimY = dimXYZ[1];
    config.dimZ = dimXYZ[2];
    config.DSIZE = config.dimX * config.dimY * config.dimZ;
    config.SSIZE = config.dimY * config.dimZ * config.dimT;

    // Allocate arrays for data storage
    float *inputArray = new float[config.DSIZE];
    float *radArray = new float[config.DSIZE];
    float *saveSlice = new float[config.SSIZE];
    float *saveActivity = new float[config.SSIZE];

    float inside = 1;  // Concentration inside
    float outside = 1; // Concentration outside
    float radVal = 0;  // Initial radical concentration

    // Initialize the first array
    for (int x = 0; x < config.dimX; x++)
    {
        for (int y = 0; y < config.dimY; y++)
        {
            for (int z = 0; z < config.dimZ; z++)
            {
                int index = x + config.dimX * (y + z * config.dimY); // Location in flat array
                if (x > 0 && y > 0 && z > 0 && x < config.dimX - 1 && y < config.dimY - 1 && z < config.dimZ - 1)
                {
                    // inside
                    inputArray[index] = inside;
                }
                else
                {
                    // outside
                    inputArray[index] = outside;
                }
                radArray[index] = radVal;
            }
        }
    }

    // Run the algorithm
    run(inputArray, radArray, saveSlice, saveActivity, config);

    // Store the data in a binary file
    // This can be opened in python with:
    // np.fromfile("data.dat", dtype=np.float32)
    // data = np.reshape(data,(10000,100,100))
    // Shape is (t, y, x)
    FILE *data = fopen("oxygenConc.dat", "wb");
    fwrite(saveSlice, sizeof(float), config.SSIZE, data);
    fclose(data);

    FILE *activity = fopen("activity.dat", "wb");
    fwrite(saveActivity, sizeof(float), config.SSIZE, activity);
    fclose(activity);

    // Free memory
    free(inputArray);
    free(radArray);
    free(saveSlice);
    free(saveActivity);

    return (0);
}
